
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define PRECISION 0.00001
#define TAM_BLOCO 4
#define uN 5.0
#define uS 5.0
#define uW 0.0
#define uE 10.0

//Variáveis GPU
__constant__ double omega = 1.5;
__constant__ double d_h1, d_h2;
__constant__ double d_denominador1, d_denominador2;
__constant__ int d_dimensaoX, d_dimensaoY;
__constant__ double d_parcial1, d_parcial2;


//Variáveis CPU
double h_h1, h_h2;
double h_denominador1, h_denominador2;
double *h_m, *d_m;
double h_parcial1, h_parcial2;
int h_dimensaoX, h_dimensaoY, laps = 0, i;


FILE *arquivo;

 clock_t start, end;
 double tempo;

//Funções da CPU

//Funcao que imprime a matriz no arquivo de saida
void printMat(){
	int i, j;
	for(i = 0; i < h_dimensaoX; i++){
		for(j = 0; j < h_dimensaoY; j++){
			fprintf(arquivo, "%lf", h_m[i * h_dimensaoY + j]);
			if(j != h_dimensaoY - 1) fprintf(arquivo, " ");
		}
		if(i != h_dimensaoX - 1)
			fprintf(arquivo, "\n");
	}
}

//Funcao que inicializa a matriz com os valores de contorno especificados pelo problema
void setupM(){
	int i,j;
	for(i = 0; i < h_dimensaoX; i++){
		for(j = 0; j < h_dimensaoY; j++){
			if(i == 0){
				h_m[i * h_dimensaoY + j] = uN;
			}else if(i == (h_dimensaoX - 1)){
				h_m[i * h_dimensaoY + j] = uS;
			}else if(j == 0){
				h_m[i * h_dimensaoY + j] = uW;
			}else if(j == h_dimensaoY - 1){
				h_m[i * h_dimensaoY + j] = uE;
			}
			else{
				h_m[i * h_dimensaoY + j] = 0;
			}
		}
	}
}

//Funções da GPU

//Funcoes "a" e "b" especificada pelo problema
__device__ double a(int i, int j){
	double x = i * d_h1;
	double y = j * d_h2;
	return 500 * x * (1 - x) * (0.5 - y);
}

__device__ double b(int i, int j){
	double x = i * d_h1;
	double y = j * d_h2;
	return 500 * y * (1 - y) * (x - 0.5);
}


//Funcoes "n", "s", "w", "e" especificadas pelo problema
__device__ double n(int i, int j){
	return (d_parcial2 - (d_h2 * b(i,j))/d_denominador2);
}
__device__ double s(int i, int j){
	return (d_parcial2 + (d_h2 * b(i,j))/d_denominador2);
}
__device__ double e(int i, int j){
	return (d_parcial1 - (d_h1 * a(i,j))/d_denominador1);
}
__device__ double w(int i, int j){
	return (d_parcial1 + (d_h1 * a(i,j))/d_denominador1);
}


//Funcao que faz a media ponderada dos valores vizinhos ao ponto que está sendo atualizado
__device__ double somaDosPontosVizinhos(int local_i, int local_j, int global_i, int global_j, float *m){

	double temp = 0;

	int dimensaoY_local = TAM_BLOCO + 2;

	temp += w(global_i, global_j) * m[(local_i - 1) * dimensaoY_local + local_j];
	temp += e(global_i, global_j) * m[(local_i + 1) * dimensaoY_local + local_j];
	temp += s(global_i, global_j) * m[local_i * dimensaoY_local + (local_j - 1)];
	temp += n(global_i, global_j) * m[local_i * dimensaoY_local + (local_j + 1)];

	// return temp;
	// if(temp > 0){
	// 	return -1;
	// }
	// if(temp == 0){
		return m[(local_i - 1) * dimensaoY_local + local_j] + m[(local_i + 1) * dimensaoY_local + local_j] + m[local_i * dimensaoY_local + (local_j - 1)] + m[local_i * dimensaoY_local + (local_j + 1)];
	// }
	// return 20;
}
__device__ double somaDosPontosVizinhos2(int i, int j, double *m){

	double temp = 0;

	temp += w(i,j) * m[(i - 1) * d_dimensaoY + j];
	temp += e(i,j) * m[(i + 1) * d_dimensaoY + j];
	temp += s(i,j) * m[i * d_dimensaoY + (j - 1)];
	temp += n(i,j) * m[i * d_dimensaoY + (j + 1)];

	return temp;
}
//Kernels principais do programa. Cada um trabalho em um conjunto de pontos da matriz
//fazendo uma media ponderada entre o valor atual do ponto que está sendo analisado e 
//seus quatro pontos adjacentes. O quanto cada valor vai pesar é determinado pelo ômega
//da funcao que, nesse caso, é fixo
__global__ void vermelhos(double *m){
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	int i_bloco = threadIdx.x + 1;
	int j_bloco = threadIdx.y + 1;

	int count = 0;
	
	__shared__ float shared_matrix[(TAM_BLOCO + 2)*(TAM_BLOCO + 2)];

	if(1){
		for(int loop = 0; loop < (TAM_BLOCO + 2)*(TAM_BLOCO+2); loop++){
			shared_matrix[loop] = 4;
		}
		for(int loop = 0; loop < (TAM_BLOCO + 2)*(TAM_BLOCO+2); loop++){
			count += shared_matrix[loop];
		}
		printf("Count %d ", count);
	}

	if(tidx != 0 && tidx < d_dimensaoX - 1 && tidy != 0 && tidy < d_dimensaoY - 1){
		// if((tidx + tidy) % 2 == 1){
			if(i_bloco == 1){
				shared_matrix[j_bloco] = m[(tidx-1) * d_dimensaoY + tidy];
			}
			if(i_bloco == TAM_BLOCO){
				shared_matrix[(TAM_BLOCO+1)* (TAM_BLOCO + 2) + j_bloco] = m[(tidx+1) * d_dimensaoY + tidy];
			}
			if(j_bloco == 1){
				shared_matrix[(i_bloco+1) * (TAM_BLOCO + 2)] = m[(tidx+1) * d_dimensaoY + tidy];
			}
			if(j_bloco == TAM_BLOCO){
				shared_matrix[(i_bloco+1) * (TAM_BLOCO + 2) + TAM_BLOCO+1] = m[(tidx+1) * d_dimensaoY + tidy + 1];
			}
			shared_matrix[i_bloco * (TAM_BLOCO + 2) + j_bloco] = m[tidx * d_dimensaoY + tidy];
			shared_matrix[i_bloco * (TAM_BLOCO + 2) + j_bloco] = m[tidx * d_dimensaoY + tidy];
		// }
		__syncthreads();

		if((tidx + tidy) % 2 == 0){			
			m[tidx * d_dimensaoY + tidy] *= (1 - omega);
			// m[tidx * d_dimensaoY + tidy] = omega * somaDosPontosVizinhos(i_bloco, j_bloco, tidx, tidy, shared_matrix);
			m[tidx * d_dimensaoY + tidy] = count;
			// m[tidx * d_dimensaoY + tidy] = 30;
		}
	}
}

__global__ void azuis(double *m){
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	int i_bloco = threadIdx.x + 1;
	int j_bloco = threadIdx.y + 1;
	
	__shared__ float shared_matrix[(TAM_BLOCO + 2)*(TAM_BLOCO + 2)];

	if(tidx != 0 && tidx < d_dimensaoX - 1 && tidy != 0 && tidy < d_dimensaoY - 1){
		if((tidx + tidy) % 2 == 0){
			// if(i_bloco == 1){
			// 	shared_matrix[j_bloco] = m[(tidx-1) * d_dimensaoY + tidy];
			// }
			// if(i_bloco == TAM_BLOCO){
			// 	shared_matrix[(TAM_BLOCO+1)* (TAM_BLOCO + 2) + j_bloco] = m[(tidx+1) * d_dimensaoY + tidy];
			// }
			// if(j_bloco == 1){
			// 	shared_matrix[i_bloco * (TAM_BLOCO + 2)] = m[tidx * d_dimensaoY + tidy -1];
			// }
			// if(j_bloco == TAM_BLOCO){
			// 	shared_matrix[i_bloco * (TAM_BLOCO + 2) + TAM_BLOCO+1] = m[tidx * d_dimensaoY + tidy + 1];
			// }
			// shared_matrix[i_bloco * (TAM_BLOCO + 2) + j_bloco] = m[tidx * d_dimensaoY + tidy];
			// shared_matrix[i_bloco * (TAM_BLOCO + 2) + j_bloco] = m[tidx * d_dimensaoY + tidy];
		}
		__syncthreads();

		if((tidx + tidy) % 2 == 1){			
			m[tidx * d_dimensaoY + tidy] *= (1 - omega);
			m[tidx * d_dimensaoY + tidy] += omega * somaDosPontosVizinhos(i_bloco, j_bloco, tidx, tidy, shared_matrix);
			// m[tidx * d_dimensaoY + tidy] = shared_matrix[i_bloco * (TAM_BLOCO + 2) + j_bloco];
		}
	}
}

int main(int argc, char** argv){

	//Especificacoes iniciais para garantir que o programa será rodado com as 
	//condicoes iniciais corretas
	if(argc != 4){
		printf("Número incorreto de parâmetros:\n");
		printf("Insira as dimensoes e a quantidade de iterações\n");
		printf("\tUtilize o formato: %s <Dimensao X> <Dimensao Y> <Iterações>\n", argv[0]);
		exit(-1);
	}

	//Inicializando todos os valores necessários para transferir para a GPU e para realizar 
	//os calculos do programa
	h_dimensaoX = atoi(argv[1]);
	h_dimensaoY = atoi(argv[2]);
	laps = atoi(argv[3]); 

	h_h1 = 1.0/(h_dimensaoX + 1);
	h_h2 = 1.0/(h_dimensaoY + 1);

	h_dimensaoX += 2;
	h_dimensaoY += 2;

	h_denominador1 = 4*(1 + (pow(h_h1,2)/pow(h_h2,2)));
	h_denominador2 = 4*(1 + (pow(h_h2,2)/pow(h_h1,2)));

	h_parcial1 = 2/h_denominador1;
	h_parcial2 = 2/h_denominador2;

	//Alocando a matriz na CPU e inicializando
	h_m = (double *) calloc(h_dimensaoX * h_dimensaoY, sizeof(double));
	setupM();

	//Alocando a matriz na GPU
	hipMalloc(&d_m, h_dimensaoX * h_dimensaoY * sizeof(double));

	//Transferindo as informações necessárias para a GPU
	hipMemcpy(d_m, h_m, h_dimensaoX * h_dimensaoY * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_denominador1), &h_denominador1, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_denominador2), &h_denominador2, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_dimensaoX), &h_dimensaoX, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_dimensaoY), &h_dimensaoY, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_h1), &h_h1, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_h2), &h_h2, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_parcial1), &h_parcial1, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_parcial2), &h_parcial2, sizeof(double), 0, hipMemcpyHostToDevice);

	//Iniciando a contagem do tempo
	start = clock();

	//Calculando a quantidade de blocos e threads que serao lancados
	dim3 nthreads(TAM_BLOCO,TAM_BLOCO);
	dim3 nblocos((h_dimensaoX + nthreads.x - 1)/nthreads.x, (h_dimensaoY + nthreads.y - 1)/nthreads.y);

	//Fazendo os cálculos
	for(i = 0; i < laps; i++){
		vermelhos<<<nblocos, nthreads>>>(d_m);
		// azuis<<<nblocos, nthreads>>>(d_m);
	}

	//Trazendo a matriz de volta para a CPU
	hipMemcpy(h_m, d_m, h_dimensaoX * h_dimensaoY * sizeof(double), hipMemcpyDeviceToHost);

	//Reseta a GPU para liberar todos os recursos
	hipDeviceReset();

	//Imprimindo a matriz no arquivo e fechando-o
	arquivo = fopen("sample.txt", "w");
	printMat();
	fclose(arquivo);

	//Termina de calcular o tempo que demorou o programa
	end = clock();
	tempo = ((double)  (end - start))/CLOCKS_PER_SEC;
	printf("Tempo total: %lfs...\n", tempo);

	return 0;
}