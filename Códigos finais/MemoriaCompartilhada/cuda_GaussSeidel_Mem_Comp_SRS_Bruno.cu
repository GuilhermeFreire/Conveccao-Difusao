
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define PRECISION 0.00001
#define TAM_BLOCO 32
#define uN 5.0
#define uS 5.0
#define uW 0.0
#define uE 10.0

//Variáveis GPU
__constant__ double omega = 1.5;
__constant__ double d_h1, d_h2;
__constant__ double d_denominador1, d_denominador2;
__constant__ int d_dimensaoX, d_dimensaoY;
__constant__ double d_parcial1, d_parcial2;

//Variáveis CPU
double h_h1, h_h2;
double h_denominador1, h_denominador2;
double *h_m, *d_m;
double h_parcial1, h_parcial2;
int h_dimensaoX, h_dimensaoY, laps = 0, i;

__device__ __shared__ double shared_matrix[TAM_BLOCO*TAM_BLOCO];


FILE *arquivo;

 clock_t start, end;
 double tempo;

//Funções da CPU

//Funcao que imprime a matriz no arquivo de saida
void printMat(){
	int i, j;
	for(i = 0; i < h_dimensaoX; i++){
		for(j = 0; j < h_dimensaoY; j++){
			fprintf(arquivo, "%lf", h_m[i * h_dimensaoY + j]);
			if(j != h_dimensaoY - 1) fprintf(arquivo, " ");
		}
		if(i != h_dimensaoX - 1)
			fprintf(arquivo, "\n");
	}
}

//Funcao que inicializa a matriz com os valores de contorno especificados pelo problema
void setupM(){
	int i,j;
	for(i = 0; i < h_dimensaoX; i++){
		for(j = 0; j < h_dimensaoY; j++){
			if(i == 0){
				h_m[i * h_dimensaoY + j] = uN;
			}else if(i == (h_dimensaoX - 1)){
				h_m[i * h_dimensaoY + j] = uS;
			}else if(j == 0){
				h_m[i * h_dimensaoY + j] = uW;
			}else if(j == h_dimensaoY - 1){
				h_m[i * h_dimensaoY + j] = uE;
			}
		}
	}
}

//Funções da GPU

//Funcoes "a" e "b" especificada pelo problema
__device__ double a(int i, int j){
	double x = i * d_h1;
	double y = j * d_h2;
	return 500 * x * (1 - x) * (0.5 - y);
}

__device__ double b(int i, int j){
	double x = i * d_h1;
	double y = j * d_h2;
	return 500 * y * (y - 1) * (x - 0.5);
}


//Funcoes "n", "s", "w", "e" especificadas pelo problema
__device__ double n(int i, int j){
	return (d_parcial2 - (d_h2 * b(i,j))/d_denominador2);
}
__device__ double s(int i, int j){
	return (d_parcial2 + (d_h2 * b(i,j))/d_denominador2);
}
__device__ double e(int i, int j){
	return (d_parcial1 - (d_h1 * a(i,j))/d_denominador1);
}
__device__ double w(int i, int j){
	return (d_parcial1 + (d_h1 * a(i,j))/d_denominador1);
}


__device__ double pontosInternos(int i_local, int j_local, int i, int j, double *m){

	double temp = 0;

	temp += w(i,j) * shared_matrix[(i_local - 1) * TAM_BLOCO + j_local];
	temp += e(i,j) * shared_matrix[(i_local + 1) * TAM_BLOCO + j_local];
	temp += s(i,j) * shared_matrix[i_local * TAM_BLOCO + (j_local - 1)];
	temp += n(i,j) * shared_matrix[i_local * TAM_BLOCO + (j_local + 1)];

	return temp;
}

__device__ double pontosExternos(int i, int j, double *m){

	double temp = 0;

	temp += w(i,j) * m[(i - 1) * d_dimensaoY + j];
	temp += e(i,j) * m[(i + 1) * d_dimensaoY + j];
	temp += s(i,j) * m[i * d_dimensaoY + (j - 1)];
	temp += n(i,j) * m[i * d_dimensaoY + (j + 1)];

	return temp;
}
//Kernels principais do programa. Cada um trabalho em um conjunto de pontos da matriz
//fazendo uma media ponderada entre o valor atual do ponto que está sendo analisado e 
//seus quatro pontos adjacentes. O quanto cada valor vai pesar é determinado pelo ômega
//da funcao que, nesse caso, é fixo
__global__ void vermelhos(double *m){
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	// if(tidx =7 && tidy == 85){
	// 	printf("%d %d\n", threadIdx.x, threadIdx.y);
	// }

	int i_bloco = threadIdx.x;
	int j_bloco = threadIdx.y;

	//Restringindo as threads ao tamanho da matriz
	if(tidx > 0 && tidy > 0 && tidx < d_dimensaoX - 1 && tidy < d_dimensaoY - 1){


			//Se for azul traz seu valor pra memória compartilhada
			if((i_bloco + j_bloco)%2 == 1){
				shared_matrix[i_bloco * TAM_BLOCO + j_bloco] = m[tidx * d_dimensaoY + tidy];
			}
			__syncthreads();

			//Vê se é um ponto externo ou interno e calcula seu valor de acordo
			if((i_bloco + j_bloco)%2 == 0){
				if(threadIdx.x > 0 && threadIdx.x < TAM_BLOCO - 2 && threadIdx.y > 0 && threadIdx.y < TAM_BLOCO - 2){
					m[tidx * d_dimensaoY + tidy] *= (1 - omega);
			 		m[tidx * d_dimensaoY + tidy] += omega * pontosInternos(i_bloco, j_bloco, tidx, tidy, m);
				}else{
					m[tidx * d_dimensaoY + tidy] *= (1 - omega);
			 		m[tidx * d_dimensaoY + tidy] += omega * pontosExternos(tidx, tidy, m);
				}
			}
	}
}

__global__ void azuis(double *m){
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	int i_bloco = threadIdx.x;
	int j_bloco = threadIdx.y;

	if(tidx > 0 && tidy > 0 && tidx < d_dimensaoX - 1 && tidy < d_dimensaoY - 1){


		if(tidx < d_dimensaoX && tidy < d_dimensaoY){

				shared_matrix[i_bloco * TAM_BLOCO + j_bloco] = m[tidx * d_dimensaoY + tidy];
			}
		

			__syncthreads();

			if((i_bloco + j_bloco)%2 == 1){
				if(threadIdx.x > 0 && threadIdx.x < TAM_BLOCO - 2 && threadIdx.y > 0 && threadIdx.y < TAM_BLOCO - 2){
					m[tidx * d_dimensaoY + tidy] *= (1 - omega);
			 		m[tidx * d_dimensaoY + tidy] += omega * pontosInternos(i_bloco, j_bloco, tidx, tidy, m);
				}else{
					m[tidx * d_dimensaoY + tidy] *= (1 - omega);
			 		m[tidx * d_dimensaoY + tidy] += omega * pontosExternos(tidx, tidy, m);
				}
			}
	}
}

int main(int argc, char** argv){

	hipDeviceReset();

	//Especificacoes iniciais para garantir que o programa será rodado com as 
	//condicoes iniciais corretas
	if(argc != 4){
		printf("Número incorreto de parâmetros:\n");
		printf("Insira as dimensoes e a quantidade de iterações\n");
 		exit(-1);
	}

	//Inicializando todos os valores necessários para transferir para a GPU e para realizar 
	//os calculos do programa
	h_dimensaoX = atoi(argv[1]);
	h_dimensaoY = atoi(argv[2]);
	laps = atoi(argv[3]); 

	h_h1 = 1.0/(h_dimensaoX + 1);
	h_h2 = 1.0/(h_dimensaoY + 1);

	h_dimensaoX += 2;
	h_dimensaoY += 2;

	h_denominador1 = 4*(1 + (pow(h_h1,2)/pow(h_h2,2)));
	h_denominador2 = 4*(1 + (pow(h_h2,2)/pow(h_h1,2)));

	h_parcial1 = 2/h_denominador1;
	h_parcial2 = 2/h_denominador2;

	//Alocando a matriz na CPU e inicializando
	h_m = (double *) calloc(h_dimensaoX * h_dimensaoY, sizeof(double));
	setupM();

	//Alocando a matriz na GPU
	hipMalloc(&d_m, h_dimensaoX * h_dimensaoY * sizeof(double));

	//Transferindo as informações necessárias para a GPU
	hipMemcpy(d_m, h_m, h_dimensaoX * h_dimensaoY * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_denominador1), &h_denominador1, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_denominador2), &h_denominador2, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_dimensaoX), &h_dimensaoX, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_dimensaoY), &h_dimensaoY, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_h1), &h_h1, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_h2), &h_h2, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_parcial1), &h_parcial1, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_parcial2), &h_parcial2, sizeof(double), 0, hipMemcpyHostToDevice);

	//Iniciando a contagem do tempo
	start = clock();

	//Calculando a quantidade de blocos e threads que serao lancados
	dim3 nthreads(TAM_BLOCO,TAM_BLOCO);
	dim3 nblocos((h_dimensaoX + nthreads.x - 1)/nthreads.x, (h_dimensaoY + nthreads.y - 1)/nthreads.y);

	printf("%d %d\n", nblocos.x , nblocos.y);
	// int j;
	// for(i = 0; i < h_dimensaoX; i++){
	// 	for(j = 0; j < h_dimensaoY; j++){
	// 		printf("%lf ", h_m[i *h_dimensaoY +j]);
	// 	}
	// 	printf("\n");
	// }
	// printf("\n");

	//Fazendo os cálculos
	for(i = 0; i < laps; i++){

		vermelhos<<<nblocos, nthreads>>>(d_m);
		//cudaDeviceSynchronize();
		gpuErrchk( hipPeekAtLastError() );
		// if(laps == 50){
		 	
		// }
		azuis<<<nblocos, nthreads>>>(d_m);
		//cudaDeviceSynchronize();
		//printf("oi %d\n",i);
		//gpuErrchk( cudaPeekAtLastError() );

	}

	//Trazendo a matriz de volta para a CPU
	hipMemcpy(h_m, d_m, h_dimensaoX * h_dimensaoY * sizeof(double), hipMemcpyDeviceToHost);

	//Reseta a GPU para liberar todos os recursos
	hipDeviceReset();

	//Imprimindo a matriz no arquivo e fechando-o
	arquivo = fopen("sample.txt", "w+");
	printMat();
	fclose(arquivo);

	//Termina de calcular o tempo que demorou o programa
	end = clock();
	tempo = ((double)  (end - start))/CLOCKS_PER_SEC;
	printf("Tempo total: %lfs...\n", tempo);

	return 0;
}
