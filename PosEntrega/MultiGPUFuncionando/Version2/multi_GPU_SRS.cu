
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define PRECISION 0.00001
#define TAM_BLOCO 8
#define uN 5.0
#define uS 5.0
#define uW 0.0
#define uE 10.0
#define GPU_ZERO 0
#define GPU_ONE 1

//Variáveis CPU
double h_h1, h_h2;
double h_denominador1, h_denominador2;
double *h0_m, *d0_m, *h1_m, *d1_m;
double *h0_auxArray, *d0_auxArray, *h1_auxArray, *d1_auxArray; 
double h_parcial1, h_parcial2;
int h_dimensaoX, h_dimensaoY, laps = 0, i, j;

//Variáveis GPU
__constant__ double omega = 1.5;
__constant__ double d_h1, d_h2;
__constant__ double d_denominador1, d_denominador2;
__constant__ int d_dimensaoX, d_dimensaoY;
__constant__ double d_parcial1, d_parcial2;

FILE *arquivo;

 clock_t start, end;
 double tempo;

//Funções da CPU

//Funcao que imprime a matriz no arquivo de saida
void printMat(){
	int i, j;
	for(i = 0; i < h_dimensaoX; i++){
		for(j = 0; j < h_dimensaoY; j++){

			if(j > h_dimensaoY*0.25){
				fprintf(arquivo, "%lf", h0_m[i * h_dimensaoY + j]);
				if(j != h_dimensaoY - 1) fprintf(arquivo, " ");
			}else{
				fprintf(arquivo, "%lf", h1_m[i * h_dimensaoY + j]);
				if(j != h_dimensaoY - 1) fprintf(arquivo, " ");
			}
		}
		if(i != h_dimensaoX - 1)
			fprintf(arquivo, "\n");
	}
}

//Funcao que inicializa a matriz com os valores de contorno especificados pelo problema
void setupM(){
	int i,j;
	for(i = 0; i < h_dimensaoX; i++){
		for(j = 0; j < h_dimensaoY; j++){
			if(i == 0){
				h1_m[i * h_dimensaoY + j] = uN;
			}else if(i == (h_dimensaoX - 1)){
				h1_m[i * h_dimensaoY + j] = uS;
			}else if(j == 0){
				h1_m[i * h_dimensaoY + j] = uW;
			}else if(j == h_dimensaoY - 1){
				h1_m[i * h_dimensaoY + j] = uE;
			}
		}
	}
}

//Funções da GPU

//Funcoes "a" e "b" especificada pelo problema
__device__ double a(int i, int j){
	double x = i * d_h1;
	double y = j * d_h2;
	return 500 * x * (1 - x) * (0.5 - y);
}

__device__ double b(int i, int j){
	double x = i * d_h1;
	double y = j * d_h2;
	return 500 * y * (1 - y) * (x - 0.5);
}


//Funcoes "n", "s", "w", "e" especificadas pelo problema
__device__ double n(int i, int j){
	return (d_parcial2 - (d_h2 * b(i,j))/d_denominador2);
}
__device__ double s(int i, int j){
	return (d_parcial2 + (d_h2 * b(i,j))/d_denominador2);
}
__device__ double e(int i, int j){
	return (d_parcial1 - (d_h1 * a(i,j))/d_denominador1);
}
__device__ double w(int i, int j){
	return (d_parcial1 + (d_h1 * a(i,j))/d_denominador1);
}


//Funcao que faz a media ponderada dos valores vizinhos ao ponto que está sendo atualizado
__device__ double somaDosPontosVizinhos(int i, int j, double *m){

	double temp = 0;

	temp += w(i,j) * m[(i - 1) * d_dimensaoY + j];
	temp += e(i,j) * m[(i + 1) * d_dimensaoY + j];
	temp += s(i,j) * m[i * d_dimensaoY + (j - 1)];
	temp += n(i,j) * m[i * d_dimensaoY + (j + 1)];

	return temp;
}

//Kernels principais do programa. Cada um trabalho em um conjunto de pontos da matriz
//fazendo uma media ponderada entre o valor atual do ponto que está sendo analisado e 
//seus quatro pontos adjacentes. O quanto cada valor vai pesar é determinado pelo ômega
//da funcao que, nesse caso, é fixo
__global__ void vermelhos(double *m, int device){
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	if(device == 1){
		if(tidx != 0 && tidx < d_dimensaoX - 1 && tidy != 0 && tidy < (d_dimensaoY - 1)*0.25){
			if((tidx + tidy) % 2 == 0){
				m[tidx * d_dimensaoY + tidy] *= (1 - omega);
				m[tidx * d_dimensaoY + tidy] += omega * somaDosPontosVizinhos(tidx, tidy, m);
			}
		}
	}else{
		if(tidx != 0 && tidx < d_dimensaoX - 1 && tidy >= (d_dimensaoY - 1)*0.25 && tidy < d_dimensaoY - 1){
			if((tidx + tidy) % 2 == 0){
				m[tidx * d_dimensaoY + tidy] *= (1 - omega);
				m[tidx * d_dimensaoY + tidy] += omega * somaDosPontosVizinhos(tidx, tidy, m);
			}
		}
	}
}

__global__ void azuis(double *m, int device){
	int tidx = blockIdx.x * blockDim.x + threadIdx.x;
	int tidy = blockIdx.y * blockDim.y + threadIdx.y;

	if(device == 1){
		if(tidx != 0 && tidx < d_dimensaoX - 1 && tidy != 0 && tidy < (d_dimensaoY - 1)*0.25){
			if((tidx + tidy) % 2 == 1){
				m[tidx * d_dimensaoY + tidy] *= (1 - omega);
				m[tidx * d_dimensaoY + tidy] += omega * somaDosPontosVizinhos(tidx, tidy, m);
			}
		}
	}else{
		if(tidx != 0 && tidx < d_dimensaoX - 1 && tidy >= (d_dimensaoY - 1)*0.25 && tidy < d_dimensaoY - 1){
			if((tidx + tidy) % 2 == 1){
				m[tidx * d_dimensaoY + tidy] *= (1 - omega);
				m[tidx * d_dimensaoY + tidy] += omega * somaDosPontosVizinhos(tidx, tidy, m);
			}
		}
	}
}

int main(int argc, char** argv){

	//Especificacoes iniciais para garantir que o programa será rodado com as 
	//condicoes iniciais corretas
	if(argc != 4){
		printf("Número incorreto de parâmetros:\n");
		printf("Insira as dimensoes e a quantidade de iterações\n");
		printf("\tUtilize o formato: %s <Dimensao X> <Dimensao Y> <Iterações>\n", argv[0]);
		exit(-1);
	}

	//Inicializando todos os valores necessários para transferir para a GPU e para realizar 
	//os calculos do programa
	h_dimensaoX = atoi(argv[1]);
	h_dimensaoY = atoi(argv[2]);
	laps = atoi(argv[3]); 

	h_h1 = 1.0/(h_dimensaoX + 1);
	h_h2 = 1.0/(h_dimensaoY + 1);

	h_dimensaoX += 2;
	h_dimensaoY += 2;

	h_denominador1 = 4*(1 + (pow(h_h1,2)/pow(h_h2,2)));
	h_denominador2 = 4*(1 + (pow(h_h2,2)/pow(h_h1,2)));

	h_parcial1 = 2/h_denominador1;
	h_parcial2 = 2/h_denominador2;

	//Alocando a matriz na CPU e inicializando
	h0_m = (double *) calloc(h_dimensaoX * h_dimensaoY, sizeof(double));
	h1_m = (double *) calloc(h_dimensaoX * h_dimensaoY, sizeof(double));
	setupM();

	//Escolhendo a GPU 0 para transferir dados
	hipSetDevice(GPU_ZERO);

	//Alocando a matriz na GPU 0
	hipMalloc(&d0_m, h_dimensaoX * h_dimensaoY * sizeof(double));

	//Transferindo as informações necessárias para a GPU 0
	hipMemcpy(d0_m, h1_m, h_dimensaoX * h_dimensaoY * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_denominador1), &h_denominador1, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_denominador2), &h_denominador2, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_dimensaoX), &h_dimensaoX, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_dimensaoY), &h_dimensaoY, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_h1), &h_h1, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_h2), &h_h2, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_parcial1), &h_parcial1, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_parcial2), &h_parcial2, sizeof(double), 0, hipMemcpyHostToDevice);


	//Escolhendo a GPU 1 para transferir dados
	hipSetDevice(GPU_ONE);

	//Alocando a matriz na GPU 1
	hipMalloc(&d1_m, h_dimensaoX * h_dimensaoY * sizeof(double));

	//Transferindo as informações necessárias para a GPU 1
	hipMemcpy(d1_m, h1_m, h_dimensaoX * h_dimensaoY * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_denominador1), &h_denominador1, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_denominador2), &h_denominador2, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_dimensaoX), &h_dimensaoX, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_dimensaoY), &h_dimensaoY, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_h1), &h_h1, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_h2), &h_h2, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_parcial1), &h_parcial1, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_parcial2), &h_parcial2, sizeof(double), 0, hipMemcpyHostToDevice);

	//Iniciando a contagem do tempo
	start = clock();

	//Calculando a quantidade de blocos e threads que serao lancados
	dim3 nthreads(TAM_BLOCO,TAM_BLOCO);
	dim3 nblocos((h_dimensaoX + nthreads.x - 1)/nthreads.x, (h_dimensaoY + nthreads.y - 1)/nthreads.y);

	//Fazendo os cálculos
	for(i = 0; i < laps; i++){

		hipSetDevice(GPU_ZERO);
		vermelhos<<<nblocos, nthreads>>>(d0_m, GPU_ZERO);
		azuis<<<nblocos, nthreads>>>(d0_m, GPU_ZERO);
		//gpuErrchk( cudaPeekAtLastError() );
		
		hipSetDevice(GPU_ONE);
		vermelhos<<<nblocos, nthreads>>>(d1_m, GPU_ONE);
		azuis<<<nblocos, nthreads>>>(d1_m, GPU_ONE);
	}

	hipSetDevice(GPU_ZERO);
	//Trazendo a matriz de volta para a CPU
	hipMemcpy(h0_m, d0_m, h_dimensaoX * h_dimensaoY * sizeof(double), hipMemcpyDeviceToHost);
	//Reseta a GPU para liberar todos os recursos
	hipDeviceReset();

	hipSetDevice(GPU_ONE);
	//Trazendo a matriz de volta para a CPU
	hipMemcpy(h1_m, d1_m, h_dimensaoX * h_dimensaoY * sizeof(double), hipMemcpyDeviceToHost);
	//Reseta a GPU para liberar todos os recursos
	hipDeviceReset();

	//Imprimindo a matriz no arquivo e fechando-o
	arquivo = fopen("sample.txt", "w");
	printMat();
	fclose(arquivo);

	//Termina de calcular o tempo que demorou o programa
	end = clock();
	tempo = ((double)  (end - start))/CLOCKS_PER_SEC;
	printf("%lf;", tempo);

	return 0;
}
